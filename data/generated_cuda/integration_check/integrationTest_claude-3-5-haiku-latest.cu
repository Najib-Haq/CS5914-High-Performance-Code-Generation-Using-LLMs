
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloKernel() {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from CUDA thread %d!\n", threadId);
}

int main() {
    int blockSize = 4;
    int numBlocks = 2;
    
    helloKernel<<<numBlocks, blockSize>>>();
    
    hipError_t cudaResult = hipDeviceSynchronize();
    if (cudaResult != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaResult));
        return 1;
    }
    
    return 0;
}