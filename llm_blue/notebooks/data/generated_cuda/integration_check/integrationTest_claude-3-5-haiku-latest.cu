#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_kernel() {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from CUDA thread %d!\n", threadId);
}

int main() {
    int numThreads = 32;
    int numBlocks = 4;

    hello_kernel<<<numBlocks, numThreads>>>();

    // Synchronize to ensure kernel execution completes
    hipDeviceSynchronize();

    // Check for any kernel launch errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    return 0;
}