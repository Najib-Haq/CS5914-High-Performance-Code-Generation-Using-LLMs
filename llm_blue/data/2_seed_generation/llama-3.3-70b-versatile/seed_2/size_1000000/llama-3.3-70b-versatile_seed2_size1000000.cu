
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// The kernel provided by the LLM
__global__ void sumReduction(int *input, int *output, int size) {
    // Optimization strategy:
    // 1. Use a sufficient number of threads per block to fully utilize the GPU's processing capabilities.
    // 2. Implement a bank-conflict-free memory access pattern in shared memory using a sequential addressing pattern.
    // 3. Utilize warp-level primitives like __shfl_down_sync() for efficient warp-level reductions.
    // 4. Apply an early exit strategy to minimize unnecessary work when the input size is less than the number of threads per block.
    // 5. Employ a combination of warp-level and block-level reductions to minimize the number of threads participating in the final reduction phase.
    // This strategy should be effective because it reduces memory traffic, optimizes computations, and maximizes utilization of GPU resources.

    extern __shared__ int sharedData[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Early exit strategy: If the size of the input array is less than the number of threads per block,
    // only the first 'size' threads participate in the reduction.
    if (idx >= size) return;

    // Load data from global memory
    int sum = 0;
    while (idx < size) {
        sum += input[idx];
        idx += stride;
    }

    // Store loaded data in shared memory using a sequential addressing pattern to minimize bank conflicts
    sharedData[threadIdx.x] = sum;
    __syncthreads();

    // Warp-level reduction using __shfl_down_sync()
    for (int offset = 16; offset > 0; offset /= 2) {
        int neighborVal = __shfl_down_sync(0xffffffff, sharedData[threadIdx.x], offset);
        if (threadIdx.x % 32 < offset) {
            sharedData[threadIdx.x] += neighborVal;
        }
    }

    // Reduction across warps in the block
    if (threadIdx.x % 32 == 0) {
        int warpId = threadIdx.x / 32;
        for (int i = 1; i < 8; i++) { // 8 warps per block (256 threads / 32 threads per warp)
            if (warpId < i) {
                sharedData[warpId * 32] += sharedData[i * 32];
            }
        }
    }

    // Store the block's result in the output array
    if (threadIdx.x == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// Host function to perform reduction
int sumArray(int* h_input, int size) {
    int *d_input, *d_temp;
    
    // Allocate device memory
    hipMalloc((void**)&d_input, size * sizeof(int));
    
    // The size of d_temp is based on the number of blocks we'll launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_temp, blocksPerGrid * sizeof(int));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Launch kernel with the actual size parameter
    hipEventRecord(start);
    sumReduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_input, d_temp, size);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy the block results back to host
    int* h_temp = new int[blocksPerGrid];
    hipMemcpy(h_temp, d_temp, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // Finalize the reduction on CPU (sum the block results)
    int sum = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        sum += h_temp[i];
    }
    
    // Print results
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;
    
    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    delete[] h_temp;
    
    return sum;
}

int main(int argc, char** argv) {
    int size = 1024;  // Default size
    
    if (argc > 1) {
        size = atoi(argv[1]);
    }
    
    // Print size for verification
    std::cout << "Running CUDA Reduction for size: " << size << std::endl;
    
    // Allocate and initialize host array
    int* h_input = new int[size];
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  // Set all elements to 1 for easy verification
    }
    
    // Run the reduction and get the sum
    int result = sumArray(h_input, size);
    
    // Verify result (should equal the array size since all elements are 1)
    bool correct = (result == size);
    std::cout << "Result verification: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Clean up
    delete[] h_input;
    
    return 0;
}
