
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// The kernel provided by the LLM
__global__ void sumReduction(int *input, int *output, int size) {
    // Optimization Strategy:
    // 1. Use shared memory to reduce global memory accesses
    // 2. Implement a tree-based parallel reduction algorithm
    // 3. Handle non-power-of-2 array sizes with boundary checks
    // 4. Use warp-level optimizations and minimize synchronization
    
    // Declare shared memory dynamically
    extern __shared__ int sharedData[];
    
    // Global thread and block indices
    int tid = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory, with boundary protection
    sharedData[tid] = (globalIdx < size) ? input[globalIdx] : 0;
    
    // Synchronize to ensure all data is loaded
    __syncthreads();
    
    // Perform parallel reduction in shared memory
    // Use log2(blockDim.x) iterations for complete reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        
        // Synchronize after each reduction step
        __syncthreads();
    }
    
    // The first thread in the block writes the block's reduction result
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// Host function to perform reduction
int sumArray(int* h_input, int size) {
    int *d_input, *d_temp;
    
    // Allocate device memory
    hipMalloc((void**)&d_input, size * sizeof(int));
    
    // The size of d_temp is based on the number of blocks we'll launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_temp, blocksPerGrid * sizeof(int));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Launch kernel with the actual size parameter
    hipEventRecord(start);
    sumReduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_input, d_temp, size);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy the block results back to host
    int* h_temp = new int[blocksPerGrid];
    hipMemcpy(h_temp, d_temp, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // Finalize the reduction on CPU (sum the block results)
    int sum = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        sum += h_temp[i];
    }
    
    // Print results
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;
    
    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    delete[] h_temp;
    
    return sum;
}

int main(int argc, char** argv) {
    int size = 1024;  // Default size
    
    if (argc > 1) {
        size = atoi(argv[1]);
    }
    
    // Print size for verification
    std::cout << "Running CUDA Reduction for size: " << size << std::endl;
    
    // Allocate and initialize host array
    int* h_input = new int[size];
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  // Set all elements to 1 for easy verification
    }
    
    // Run the reduction and get the sum
    int result = sumArray(h_input, size);
    
    // Verify result (should equal the array size since all elements are 1)
    bool correct = (result == size);
    std::cout << "Result verification: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Clean up
    delete[] h_input;
    
    return 0;
}
