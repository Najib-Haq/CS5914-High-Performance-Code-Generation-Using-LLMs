
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// The kernel provided by the LLM
__global__ void sumReduction(int *input, int *output, int size) {
    // Optimization Strategy:
    // 1. Combine shuffle-based and shared memory reduction
    // 2. Use warp-level shuffle instructions for initial reduction
    // 3. Minimize global memory accesses and synchronization
    // 4. Handle non-power-of-2 sizes with predicated load
    // 5. Reduce bank conflicts through sequential addressing
    // 6. Leverage instruction-level parallelism with unrolled reduction
    
    // Declare shared memory for final block-level reduction
    extern __shared__ int sharedData[];
    
    // Global thread indices
    int tid = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Predicated load with boundary check
    int value = (globalIdx < size) ? input[globalIdx] : 0;
    
    // Warp-level reduction using shuffle instructions
    // Unrolled for better performance
    #pragma unroll
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        value += __shfl_down_sync(0xffffffff, value, offset);
    }
    
    // Store warp reduction result for first thread of each warp
    if (tid % warpSize == 0) {
        sharedData[tid / warpSize] = value;
    }
    
    // Block-level synchronization
    __syncthreads();
    
    // Final block reduction using shared memory
    // First warp handles final reduction
    if (tid < warpSize) {
        // Load warp reduction results, handling potential empty warps
        value = (tid < (blockDim.x + warpSize - 1) / warpSize) ? 
                sharedData[tid] : 0;
        
        // Final warp-level shuffle reduction
        #pragma unroll
        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            value += __shfl_down_sync(0xffffffff, value, offset);
        }
        
        // First thread writes block result
        if (tid == 0) {
            output[blockIdx.x] = value;
        }
    }
}

// Host function to perform reduction
int sumArray(int* h_input, int size) {
    int *d_input, *d_temp;
    
    // Allocate device memory
    hipMalloc((void**)&d_input, size * sizeof(int));
    
    // The size of d_temp is based on the number of blocks we'll launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_temp, blocksPerGrid * sizeof(int));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Launch kernel with the actual size parameter
    hipEventRecord(start);
    sumReduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_input, d_temp, size);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy the block results back to host
    int* h_temp = new int[blocksPerGrid];
    hipMemcpy(h_temp, d_temp, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // Finalize the reduction on CPU (sum the block results)
    int sum = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        sum += h_temp[i];
    }
    
    // Print results
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;
    
    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    delete[] h_temp;
    
    return sum;
}

int main(int argc, char** argv) {
    int size = 1024;  // Default size
    
    if (argc > 1) {
        size = atoi(argv[1]);
    }
    
    // Print size for verification
    std::cout << "Running CUDA Reduction for size: " << size << std::endl;
    
    // Allocate and initialize host array
    int* h_input = new int[size];
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  // Set all elements to 1 for easy verification
    }
    
    // Run the reduction and get the sum
    int result = sumArray(h_input, size);
    
    // Verify result (should equal the array size since all elements are 1)
    bool correct = (result == size);
    std::cout << "Result verification: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Clean up
    delete[] h_input;
    
    return 0;
}
