
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// The kernel provided by the LLM
/*
 * Optimization strategy:
 * 1. Uses sequential addressing to avoid bank conflicts in shared memory
 * 2. Unrolls the last warp to avoid unnecessary synchronization
 * 3. Uses template loop unrolling for reducing shared memory accesses
 * 4. Processes multiple elements per thread during load to reduce total threads needed
 * 5. Uses volatile shared memory for warp operations to ensure proper memory ordering
 */
__global__ void sumReduction(int *input, int *output, int size) {
    extern __shared__ int sdata[];
    
    // Each thread loads and adds 4 elements to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 4) + threadIdx.x;
    unsigned int gridSize = blockDim.x * 4 * gridDim.x;
    
    // Initialize shared memory with identity value
    sdata[tid] = 0;
    
    // Grid-stride loop to handle large arrays
    while (i < size) {
        int sum = 0;
        if (i < size) sum += input[i];
        if (i + blockDim.x < size) sum += input[i + blockDim.x];
        if (i + 2 * blockDim.x < size) sum += input[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < size) sum += input[i + 3 * blockDim.x];
        sdata[tid] += sum;
        i += gridSize;
    }
    __syncthreads();

    // Do reduction in shared memory
    if (blockDim.x >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    
    // Unroll last 6 iterations (warp is synchronized)
    if (tid < 32) {
        volatile int *smem = sdata;
        if (blockDim.x >= 64) smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }
    
    // Write result for this block to global memory
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

// Host function to perform reduction
int sumArray(int* h_input, int size) {
    int *d_input, *d_temp;
    
    // Allocate device memory
    hipMalloc((void**)&d_input, size * sizeof(int));
    
    // The size of d_temp is based on the number of blocks we'll launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_temp, blocksPerGrid * sizeof(int));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Launch kernel with the actual size parameter
    hipEventRecord(start);
    sumReduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_input, d_temp, size);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy the block results back to host
    int* h_temp = new int[blocksPerGrid];
    hipMemcpy(h_temp, d_temp, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // Finalize the reduction on CPU (sum the block results)
    int sum = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        sum += h_temp[i];
    }
    
    // Print results
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;
    
    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    delete[] h_temp;
    
    return sum;
}

int main(int argc, char** argv) {
    int size = 1024;  // Default size
    
    if (argc > 1) {
        size = atoi(argv[1]);
    }
    
    // Print size for verification
    std::cout << "Running CUDA Reduction for size: " << size << std::endl;
    
    // Allocate and initialize host array
    int* h_input = new int[size];
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  // Set all elements to 1 for easy verification
    }
    
    // Run the reduction and get the sum
    int result = sumArray(h_input, size);
    
    // Verify result (should equal the array size since all elements are 1)
    bool correct = (result == size);
    std::cout << "Result verification: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Clean up
    delete[] h_input;
    
    return 0;
}
