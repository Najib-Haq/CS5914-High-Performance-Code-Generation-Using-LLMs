
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// The kernel provided by the LLM
/*
 * Optimization strategy:
 * 1. Uses 16 elements per thread during load to significantly reduce memory transactions
 *    and total thread count (better for very large arrays)
 * 2. Implements two-stage reduction: first parallel within warps using shuffle,
 *    then across warps using shared memory
 * 3. Completely eliminates shared memory bank conflicts using sequential addressing
 * 4. Minimizes divergent execution by removing most conditional statements
 * 5. Uses grid-stride loop with unrolled inner loop for handling large arrays
 * 6. Employs full warp-level primitives to eliminate synchronization in final stages
 * 7. Reduces register pressure by reusing variables where possible
 * 8. Maintains coalesced memory access patterns during initial loading
 */
__global__ void sumReduction(int *input, int *output, int size) {
    extern __shared__ int sdata[];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int wid = tid >> 5;  // Warp ID
    const unsigned int lane = tid & 31;  // Lane within warp
    const unsigned int gridSize = blockDim.x * gridDim.x * 16;
    unsigned int idx = blockIdx.x * (blockDim.x * 16) + tid;
    
    // Thread-local sum
    int sum = 0;
    
    // Grid-stride loop, each thread handles 16 elements
    while (idx < size) {
        #pragma unroll
        for (int i = 0; i < 16; i++) {
            unsigned int curr_idx = idx + i * blockDim.x;
            if (curr_idx < size) {
                sum += input[curr_idx];
            }
        }
        idx += gridSize;
    }
    
    // First reduction using warp shuffle
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
    
    // Write warp results to shared memory
    if (lane == 0) {
        sdata[wid] = sum;
    }
    __syncthreads();
    
    // Final reduction across warps
    if (tid < (blockDim.x >> 5)) {  // Only first warp
        sum = (tid < (blockDim.x >> 5)) ? sdata[tid] : 0;
        
        #pragma unroll
        for (int offset = (blockDim.x >> 6); offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        
        if (lane == 0) {
            output[blockIdx.x] = sum;
        }
    }
}

// Host function to perform reduction
int sumArray(int* h_input, int size) {
    int *d_input, *d_temp;
    
    // Allocate device memory
    hipMalloc((void**)&d_input, size * sizeof(int));
    
    // The size of d_temp is based on the number of blocks we'll launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_temp, blocksPerGrid * sizeof(int));
    
    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Launch kernel with the actual size parameter
    hipEventRecord(start);
    sumReduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_input, d_temp, size);
    hipEventRecord(stop);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy the block results back to host
    int* h_temp = new int[blocksPerGrid];
    hipMemcpy(h_temp, d_temp, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
    
    // Finalize the reduction on CPU (sum the block results)
    int sum = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        sum += h_temp[i];
    }
    
    // Print results
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;
    
    // Clean up
    hipFree(d_input);
    hipFree(d_temp);
    delete[] h_temp;
    
    return sum;
}

int main(int argc, char** argv) {
    int size = 1024;  // Default size
    
    if (argc > 1) {
        size = atoi(argv[1]);
    }
    
    // Print size for verification
    std::cout << "Running CUDA Reduction for size: " << size << std::endl;
    
    // Allocate and initialize host array
    int* h_input = new int[size];
    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  // Set all elements to 1 for easy verification
    }
    
    // Run the reduction and get the sum
    int result = sumArray(h_input, size);
    
    // Verify result (should equal the array size since all elements are 1)
    bool correct = (result == size);
    std::cout << "Result verification: " << (correct ? "PASSED" : "FAILED") << std::endl;
    
    // Clean up
    delete[] h_input;
    
    return 0;
}
