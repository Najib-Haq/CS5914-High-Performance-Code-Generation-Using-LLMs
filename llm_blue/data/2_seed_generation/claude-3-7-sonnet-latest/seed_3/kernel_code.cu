
#include <hip/hip_runtime.h>
__global__ void sumReduction(int *input, int *output, int size) {
    // Optimization strategy:
    // 1. Sequential addressing to avoid bank conflicts in shared memory
    // 2. Two-element processing per thread during loading to reduce the initial data size
    // 3. Grid-stride loop to handle arbitrary sized inputs efficiently
    // 4. Complete unrolling for last 6 iterations (covers warp size of 32)
    // 5. Use warp shuffle operations for warp-level reductions to eliminate shared memory and sync in final stages
    // 6. Early exit when thread is out of bounds to avoid unnecessary work
    // 7. First add operation during loading to reduce total number of reduction steps
    // 8. Minimize divergent branches by consolidating conditionals
    // 9. Utilize full warp for shuffle operations for maximum throughput
    
    extern __shared__ int sdata[];
    
    // Thread and block identification
    unsigned int tid = threadIdx.x;
    unsigned int blockSize = blockDim.x;
    unsigned int globalIdx = blockIdx.x * blockDim.x * 2 + threadIdx.x; // *2 for loading two elements
    unsigned int gridSize = blockDim.x * gridDim.x * 2; // *2 for the stride
    
    // Each thread accumulates results into a local variable
    int thread_sum = 0;
    
    // Grid-stride loop - each thread processes 2 elements per iteration
    for (unsigned int i = globalIdx; i < size; i += gridSize) {
        thread_sum += input[i];
        
        // Check if the second element is within bounds
        if (i + blockDim.x < size) {
            thread_sum += input[i + blockDim.x];
        }
    }
    
    // Store sum in shared memory
    sdata[tid] = thread_sum;
    __syncthreads();
    
    // Sequential addressing: reduce with fewer bank conflicts and less divergence
    // Only do this for larger reductions, stop when we reach warp size
    for (unsigned int s = blockSize / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Last warp reduction using warp shuffle
    if (tid < 32) {
        // Handle the case where blockSize is not a multiple of warp size
        if (blockSize >= 64) {
            sdata[tid] += sdata[tid + 32];
        }
        
        // Use warp shuffle operations for the final reduction (no sync needed)
        // Fully unrolled for maximum performance
        int value = sdata[tid];
        
        // Unroll for all possible warp sizes to ensure the compiler selects optimal code
        value += __shfl_down_sync(0xffffffff, value, 16);
        value += __shfl_down_sync(0xffffffff, value, 8);
        value += __shfl_down_sync(0xffffffff, value, 4);
        value += __shfl_down_sync(0xffffffff, value, 2);
        value += __shfl_down_sync(0xffffffff, value, 1);
        
        // Only first thread writes result for this block
        if (tid == 0) {
            output[blockIdx.x] = value;
        }
    }
}