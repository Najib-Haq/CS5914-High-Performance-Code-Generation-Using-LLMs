#include <iostream>
#include <hip/hip_runtime.h>


__global__ void SimpleSumReductionKernel(int* input, int* output) {
    unsigned int i = 2 * threadIdx.x;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            input[i] += input[i + stride];
        }
        __syncthreads();

    }
    if (threadIdx.x == 0) {
    *output = input[0];
    }
}

int main() {
    // Size of the input data
    const int size = 1024;
    const int bytes = size * sizeof(int);

    // Allocate memory for input and output on host
    int* h_input = new int[size];
    int* h_output = new int;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1; // Example: Initialize all elements to 1
    }

    // Allocate memory for input and output on device
    int* d_input;
    int* d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, sizeof(int));

    // benchmark
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    hipEventRecord(start);
    // Launch the kernel
    SimpleSumReductionKernel<<<1, size / 2>>>(d_input, d_output);
    hipEventRecord(stop);

    // benchmark
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution Time: %f ms\n", milliseconds); 

    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Sum is " << *h_output << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}