#include <iostream>
#include <hip/hip_runtime.h>

#include <nvml.h>  // NVIDIA Management Library

#define THREADS_PER_BLOCK 1024

// Warp-Level Reduction Kernel
__inline__ __device__ int warpReduceSum(int sum) {
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }
    return sum;
}

// Fully Optimized Reduction Kernel (Warp-Level)
__global__ void warpShuffleReduceKernel(int* input, int* output, long long int size) {
    __shared__ int shared_data[THREADS_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned long long i = blockIdx.x * (blockDim.x * 2) + tid;

    // Load input safely and prevent out-of-bounds errors
    int sum = (i < size ? input[i] : 0) + (i + blockDim.x < size ? input[i + blockDim.x] : 0);
    shared_data[tid] = sum;
    __syncthreads();

    // Block-Level Reduction (Shared Memory)
    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // Warp-Level Reduction (Using Shuffle Instructions)
    if (tid < 32) {
        int val = shared_data[tid];
        // warp shuffle reduction
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }
        if (tid == 0) output[blockIdx.x] = val;
    }
}

// Reduce Partial Sums on CPU
int finalReduceOnCPU(int* d_output, int numBlocks) {
    int* h_partial_sums = new int[numBlocks];
    hipMemcpy(h_partial_sums, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

    int totalSum = 0;
    for (int i = 0; i < numBlocks; i++) {
        totalSum += h_partial_sums[i];
    }

    delete[] h_partial_sums;
    return totalSum;
}

// GPU Memory Check Function
void checkGpuMemoryUsage(const char* message) {
    nvmlInit();
    nvmlDevice_t device;
    nvmlMemory_t memory;
    nvmlDeviceGetHandleByIndex(0, &device);
    nvmlDeviceGetMemoryInfo(device, &memory);
    std::cout << message << " - GPU Memory Usage: " 
              << (memory.used / (1024 * 1024)) << " MB / "
              << (memory.total / (1024 * 1024)) << " MB\n";
    nvmlShutdown();
}

// Host Function to Call Kernel
void reduce(int* d_input, int* d_output, long long int size) {
    int numBlocks = (size + THREADS_PER_BLOCK * 2 - 1) / (THREADS_PER_BLOCK * 2);
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    warpShuffleReduceKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, size);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    int finalSum = finalReduceOnCPU(d_output, numBlocks);
    printf("Warp Shuffle Reduction Kernel Execution Time: %f ms\n", milliseconds);
    printf("Correct Final Sum: %d\n", finalSum);
}

// Main Function
int main() {
    const long long int size = 1024; // Adjustable (1K, 1M, 1B, 5B)
    const size_t bytes = size * sizeof(int);
    
    int* h_input = new int[size];
    for (long long int i = 0; i < size; i++) h_input[i] = 1;

    int *d_input, *d_output;
    checkGpuMemoryUsage("Before GPU Memory Allocation");

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, ((size + THREADS_PER_BLOCK * 2 - 1) / (THREADS_PER_BLOCK * 2)) * sizeof(int));  

    checkGpuMemoryUsage("After GPU Memory Allocation");

    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);
    checkGpuMemoryUsage("After Copying Data to GPU");

    reduce(d_input, d_output, size);

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;

    checkGpuMemoryUsage("After GPU Memory Deallocation");
    return 0;
}
