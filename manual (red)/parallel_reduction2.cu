#include <iostream>
#include <hip/hip_runtime.h>


__global__ void FixDivergenceKernel(int* input, int* output) {
    unsigned int i = threadIdx.x; //threads start next to each other
    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) { // furthest element is blockDim away
        if (threadIdx.x < stride) { // 
            input[i] += input[i + stride]; // each thread adds a distant element to its assigned position
        }
        __syncthreads();

    }
    if (threadIdx.x == 0) {
    *output = input[0];
    }
}

int main() {
    // Size of the input data
    const int size = 1024;
    const int bytes = size * sizeof(int);

    // Allocate memory for input and output on host
    int* h_input = new int[size];
    int* h_output = new int;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1; // Example: Initialize all elements to 1
    }

    // Allocate memory for input and output on device
    int* d_input;
    int* d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, sizeof(int));

    // benchmark
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    hipEventRecord(start);
    // Launch the kernel
    FixDivergenceKernel<<<1, size / 2>>>(d_input, d_output);
    hipEventRecord(stop);

    // benchmark
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution Time: %f ms\n", milliseconds); 

    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Sum is " << *h_output << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}